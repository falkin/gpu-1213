#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "fractal.h"
#include "computeFractals.h"
#include <stdint.h>
#include "DomaineMaths.h"
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include <stdio.h>

/**
 *
 */
__global__ void kernelMandelbrot ( uchar4* ptrDevImageGL, const uint32_t w, const uint32_t h, const DomainMathsCuda domain,
    const uint32_t n ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int32_t i, j;
  double x, y;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    toXY ( domain, i, j, x, y );
    displayMandelbrotXY ( x, y, n, ptrDevImageGL, s );
    s += nbThreads;
  }
}

/**
 *
 */
__global__ void kernelJulia ( uchar4* ptrDevImageGL, const uint32_t w, const uint32_t h, const DomainMathsCuda domain, const uint32_t n,
    const float real_c, const float img_c ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int32_t i, j;
  double x, y;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    toXY ( domain, i, j, x, y );
    displayJuliaXY ( x, y, n, real_c, img_c, ptrDevImageGL, s );
    s += nbThreads;
  }
}

/**
 *
 */
void launchKernelFillImageFractal ( uchar4* ptrDevImageGL, const uint32_t w, const uint32_t h, const DomaineMaths& domain,
    const FractalType type, const uint32_t n, const float real_c, const float img_c ) {
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
  DomainMathsCuda cudaDomain = createDomaineMathsCuda ( domain, w, h );
  switch ( type ) {
  case MANDELBROT:
    kernelMandelbrot<<< dg, db >>>( ptrDevImageGL, w, h, cudaDomain, n );
    break;
  case JULIA:
    kernelJulia<<< dg, db >>>( ptrDevImageGL, w, h, cudaDomain, n, real_c, img_c );
    break;
  case NEWTON:
    break;
  }
}
