#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "ripplingcompute.h"

__global__ void kernelFillImage ( uchar4* ptrDevImageGL, int w, int h, float t ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    computeRippling ( ptrDevImageGL[s], w, i, j, t );
    s += nbThreads;
  }
}

void launchKernelFillImage ( uchar4* ptrDevImageGL, int w, int h, float t ) {
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
kernelFillImage<<<dg, db>>>(ptrDevImageGL, w, h, t);
}
