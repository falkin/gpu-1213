#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "ripplingcompute.h"

__global__ void kernelFillImageRippling ( uchar4* ptrDevImageGL, int w, int h, float t ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    computeRippling ( ptrDevImageGL[s], w, i, j, t );
    s += nbThreads;
  }
}

void launchKernelFillImageRippling ( uchar4* ptrDevImageGL, int w, int h, float t ) {
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
kernelFillImageRippling<<<dg, db>>>(ptrDevImageGL, w, h, t);
}
