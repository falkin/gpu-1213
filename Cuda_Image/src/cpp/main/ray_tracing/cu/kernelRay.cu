#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "compute.h"
#include "Sphere.h"

__global__ void kernelFillImageRay ( uchar4* ptrDevImageGL, int w, int h, float t , Sphere* ptrDevSphereArray, int nbSphere) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    compute( ptrDevImageGL[s], w, i, j, t ,ptrDevSphereArray, nbSphere);
    s += nbThreads;

  }
}

void launchKernelFillImageRay ( uchar4* ptrDevImageGL, int w, int h, float t ) {
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );

  int nbSphere = 10;
  Sphere* ptrHostSphereArray = new Sphere[nbSphere];
  Sphere* ptrDevSphereArray = NULL;

  size_t arraySize = nbSphere* sizeof(Sphere);
  HANDLE_ERROR(hipMalloc((void**) &ptrDevSphereArray, arraySize));


  for (int i=0;i<nbSphere;i++)
      {
      float3 centre = {i*30,i*30,500};
      ptrHostSphereArray[i] = *(new Sphere(centre,20.0,2.0));
      }

  HANDLE_ERROR(hipMemcpy(ptrDevSphereArray,ptrHostSphereArray,arraySize,hipMemcpyHostToDevice));

kernelFillImageRay<<<dg, db>>>(ptrDevImageGL, w, h, t, ptrDevSphereArray, nbSphere);
}
