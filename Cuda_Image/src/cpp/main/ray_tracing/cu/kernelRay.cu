#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "compute.h"
#include "Sphere.h"
#include <stdio.h>



__constant__ Sphere ARRAY_DATA[MAX_SPHERE];

__device__ void copyGMtoSM ( Sphere* tabSM, Sphere* tabGM, int n );

extern __shared__ Sphere ptrShareSphereArray[];

__global__ void kernelFillImageRayShared ( uchar4* ptrDevImageGL, int w, int h, float t, Sphere* ptrDevSphereArray, int nbSphere ) {
  int tid = Indice2D::tid ();
  int tidLocal = Indice2D::tidLocalBlock ();
  int nbThreads = Indice2D::nbThread ();

  copyGMtoSM ( ptrShareSphereArray, ptrDevSphereArray, nbSphere );

  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    compute ( ptrDevImageGL[s], w, i, j, t, ptrShareSphereArray, nbSphere );
    s += nbThreads;
  }
}

__global__ void kernelFillImageRayConstant ( uchar4* ptrDevImageGL, int w, int h, float t, int nbSphere ) {
  int tid = Indice2D::tid ();
  int tidLocal = Indice2D::tidLocalBlock ();
  int nbThreads = Indice2D::nbThread ();

  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    compute ( ptrDevImageGL[s], w, i, j, t, ARRAY_DATA, nbSphere );
    s += nbThreads;
  }
}

__device__ void copyGMtoSM ( Sphere* tabSM, Sphere* tabGM, int n ) {
  int tidLocal = Indice2D::tidLocalBlock ();
  int nbThreadBlock = Indice2D::nbThreadBlock ();

  while ( tidLocal < n ) {
    tabSM[tidLocal] = tabGM[tidLocal];
    tidLocal += nbThreadBlock;
  }
  __syncthreads ();
}

__global__ void kernelFillImageRayGlobal ( uchar4* ptrDevImageGL, int w, int h, float t, Sphere* ptrDevSphereArray, int nbSphere ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    compute ( ptrDevImageGL[s], w, i, j, t, ptrDevSphereArray, nbSphere );
    s += nbThreads;
  }
}

void launchKernelFillImageRay(uchar4* ptrDevImageGL, int w, int h, float t, Sphere* ptrHostSphereArray, Sphere* ptrDevSphereArray, int nbSphere,
	MemType memType, dim3 dg, dim3 db)
    {
    switch (memType)
	{
	case GLOBAL:
	    kernelFillImageRayGlobal<<<dg, db>>>(ptrDevImageGL, w, h, t, ptrDevSphereArray, nbSphere);
	    break;
	case SHARED:
	    {
	    int nbByte = nbSphere * sizeof(Sphere);
	    kernelFillImageRayShared<<<dg, db, nbByte>>>(ptrDevImageGL, w, h, t, ptrDevSphereArray, nbSphere);
	    break;
	    }
	case CONSTANT:
	    kernelFillImageRayConstant<<<dg, db>>>(ptrDevImageGL, w, h, t, nbSphere);
	    break;
	}
}
