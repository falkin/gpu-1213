#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "ColorToolCuda.h"
#include <stdint.h>
#include <stdio.h>

/**
 *
 */
struct CalibreCuda {
  float _slope;
  float _t;
  float _x1;
  float _x2;
  float _y1;
  float _y2;
};

/**
 *
 */
CalibreCuda createCalibreCuda ( const float x1, const float x2, const float y1, const float y2 ) {
  float slope = ( y2 - y1 ) / ( x2 - x1 );
  CalibreCuda calibre = { slope, ( y1 - slope * x1 ), x1, x2, y1, y2 };
  return calibre;
}

/**
 *
 */
__device__ bool isInsideBounds ( const uint32_t borderWLeft, const uint32_t borderWRight, const uint32_t borderHUp,
    const uint32_t borderHDown, const uint32_t i, const uint32_t j ) {
  bool inside = i < borderWRight;
  inside &= i > borderWLeft;
  inside &= j < borderHDown;
  inside &= j > borderHUp;
  return inside;
}

/**
 *
 */
__device__
float calibrate ( const CalibreCuda& calibre, const float x ) {
  return ( calibre._slope * x ) + calibre._t;
}

/*--------------------------------------*\
 |*   GPU Globals                       *|
 \*-------------------------------------*/

/**
 *
 */
__device__ float sumNeighbours ( const float* ptrDevSrc, const uint32_t i, const uint32_t j, const uint32_t w, const uint32_t h ) {
  float sum = 0.0;
  sum += isInsideBounds ( 0, w, 0, h, i - 1, j ) ? ptrDevSrc[Indice2D::ijToS ( i - 1, j, w )] : 0.0f;
  sum += isInsideBounds ( 0, w, 0, h, i + 1, j ) ? ptrDevSrc[Indice2D::ijToS ( i + 1, j, w )] : 0.0f;
  sum += isInsideBounds ( 0, w, 0, h, i, j - 1 ) ? ptrDevSrc[Indice2D::ijToS ( i, j - 1, w )] : 0.0f;
  sum += isInsideBounds ( 0, w, 0, h, i, j + 1 ) ? ptrDevSrc[Indice2D::ijToS ( i, j + 1, w )] : 0.0f;
  return sum;
}

/**
 *
 */
__global__ void kernelHeatErase ( const float* ptrDevSrc, float* ptrDevDest, const uint32_t w, const uint32_t h ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    ptrDevDest[s] = ptrDevSrc[s] != 0.0f ? ptrDevSrc[s] : ptrDevDest[s];
    s += nbThreads;
  }
}

/**
 *
 */
__global__ void kernelHeatDiffuse ( const float* ptrDevSrc, float* ptrDevDest, const uint32_t w, const uint32_t h, const float k ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    float old = ptrDevSrc[s];
    float neighbours = sumNeighbours ( ptrDevSrc, i, j, w, h );
    ptrDevDest[s] = old + k * ( neighbours - ( 4 * old ) );
    s += nbThreads;
  }
}

/**
 *
 */
__global__ void kernelHeatInitArrays ( float* ptrDevArray, const uint32_t w, const uint32_t h, const float fill ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    ptrDevArray[s] = fill;
    s += nbThreads;
  }
}

/**
 *
 */
__global__ void kernelHeatInitHeaters ( float* ptrDevHeater, const uint32_t w, const uint32_t h, const float fillBig, const float fillSmall,
    const size_t bigSquare, const size_t smallSquare ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  size_t halfW = w / 2;
  size_t quarterW = w / 4;
  size_t threeQuarterW = 3 * w / 4;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    bool big = isInsideBounds ( halfW - bigSquare, halfW + bigSquare, halfW - bigSquare, halfW + bigSquare, i, j );
    bool small = isInsideBounds ( quarterW - smallSquare, quarterW + smallSquare, quarterW - smallSquare, quarterW + smallSquare, i, j ); // LU => center ( quarterW, quarterW )
    small |= isInsideBounds ( threeQuarterW - smallSquare, threeQuarterW + smallSquare, quarterW - smallSquare, quarterW + smallSquare, i,
        j ); // RU => center ( threeQuarterW, quarterW )
    small |= isInsideBounds ( quarterW - smallSquare, quarterW + smallSquare, threeQuarterW - smallSquare, threeQuarterW + smallSquare, i,
        j ); // LD => center ( quarterW, threeQuarterW )
    small |= isInsideBounds ( threeQuarterW - smallSquare, threeQuarterW + smallSquare, threeQuarterW - smallSquare,
        threeQuarterW + smallSquare, i, j ); // RD => center ( threeQuarterW, threeQuarterW )

    ptrDevHeater[s] = big ? fillBig : ( small ? fillSmall : 0.0 );
    s += nbThreads;
  }
}

/**
 *
 */
__global__ void kernelHeatDisplay ( const float* ptrDevArray, const uint32_t w, const uint32_t h, const CalibreCuda calibre,
    uchar4* ptrDevImageGL ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    float hue = calibrate ( calibre, ptrDevArray[s] );
    ColorToolCuda::HSB_TO_RVB ( hue, 1, 1, ptrDevImageGL[s].x, ptrDevImageGL[s].y, ptrDevImageGL[s].z );
    ptrDevImageGL[s].w = 255;
    s += nbThreads;
  }
}

/*--------------------------------------*\
 |*   CPU Globals                       *|
 \*-------------------------------------*/
static float* ptrDevHeater = NULL;
static float* ptrDevIniter = NULL;
static float* ptrDevArrayA = NULL;
static float* ptrDevArrayB = NULL;
static CalibreCuda calibre = createCalibreCuda ( 0, 1, 0.7, 0 );

/**
 * Initializes 4 arrays to compute heat dispersion :
 * <ul>
 * <li>heaters</li>
 * <li>initial</li>
 * <li>array A</li>
 * <li>array B</li>
 * </ul>
 *
 * Array A and B are used to diffuse heat and to display.
 *
 * @param w width of the image
 * @param h heigth of the image
 * @param k heat dispersion constant
 * @param blintItr number of blind iteration to do
 */
void initKernelHeatDispersion ( const uint32_t w, const uint32_t h, const float k, const uint32_t blindItr ) {
  // Assign memory
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
  size_t size = w * h * sizeof(float);
  HANDLE_ERROR ( hipMalloc ( (void**) &ptrDevHeater, size ) );
  HANDLE_ERROR ( hipMalloc ( (void**) &ptrDevIniter, size ) );
  HANDLE_ERROR ( hipMalloc ( (void**) &ptrDevArrayA, size ) );
  HANDLE_ERROR ( hipMalloc ( (void**) &ptrDevArrayB, size ) );

  // Fill each value with 0s
  kernelHeatInitArrays<<< dg, db >>> ( ptrDevHeater, w, h, 0.0 );
  kernelHeatInitArrays<<< dg, db >>> ( ptrDevIniter, w, h, 0.0 );
  kernelHeatInitArrays<<< dg, db >>> ( ptrDevArrayA, w, h, 0.0 );
  kernelHeatInitArrays<<< dg, db >>> ( ptrDevArrayB, w, h, 0.0 );

  // Init heaters, middle filled with 0, corners squares filled with 0.5
  size_t bigSquare = w / 10;
  size_t smallSquare = w / 20;
  kernelHeatInitHeaters<<< db, dg >>> ( ptrDevHeater, w, h, 1.0, 0.5, bigSquare, smallSquare );

  // last part of initialization, Erase -> Diffuse -> Erase
  kernelHeatErase<<< dg, db >>> ( ptrDevHeater, ptrDevIniter, w, h );
  kernelHeatDiffuse<<< dg, db >>> ( ptrDevIniter, ptrDevArrayA, w, h, k );
  kernelHeatErase<<< dg, db >>> ( ptrDevHeater, ptrDevArrayA, w, h );
  ;// Just to correct auto-formating from Eclipse
}

/**
 *
 */
void launchKernelHeatDispersion ( const uint32_t w, const uint32_t h, const float k, const uint32_t blindItr, uchar4* ptrDevImageGL ) {
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
  // a chaque tour (A -> B -> A ...)
  kernelHeatDisplay<<< db, dg >>> ( ptrDevArrayA, w, h, calibre, ptrDevImageGL );
  for ( int i = 0; i < blindItr; i++ ) {
    kernelHeatDiffuse<<< dg, db >>> ( ptrDevArrayA, ptrDevArrayB, w, h, k );
    kernelHeatErase<<< dg, db >>> ( ptrDevHeater, ptrDevArrayB, w, h );
    float* tmp = ptrDevArrayA; // swapping
    ptrDevArrayA = ptrDevArrayB;
    ptrDevArrayB = tmp;
  }

  ; // Just to correct auto-formating from Eclipse
}
