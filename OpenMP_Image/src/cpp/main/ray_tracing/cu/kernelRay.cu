#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "compute.h"
#include "Sphere.h"


__global__ void kernelFillImageRay ( uchar4* ptrDevImageGL, int w, int h, float t , Sphere* ptrDevSphereArray, int nbSphere) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  int i, j;
  while ( s < h * w ) {
    Indice2D::pixelIJ ( s, w, i, j );
    compute( ptrDevImageGL[s], w, i, j, t ,ptrDevSphereArray, nbSphere);
    s += nbThreads;
  }
}

void launchKernelFillImageRay ( uchar4* ptrDevImageGL, int w, int h, float t, Sphere* ptrHostSphereArray, Sphere* ptrDevSphereArray,int nbSphere ) {
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
kernelFillImageRay<<<dg, db>>>(ptrDevImageGL, w, h, t, ptrDevSphereArray, nbSphere);
}
