#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

void pimcCuda(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void kernelReduction(hiprandState* tabGeneratorThread, int* ptrDevTab, int n);
__global__ static void kernelGenRandom(hiprandState* tabGeneratorThread, int deviceId);
__device__ static void step(int* tab, int half);
__device__ void copyGMtoSM(int* tabSM, int* tabGM, int n);
__device__ void copySMtoGM(int* tabGM, int* tabSM, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

#define THREAD_PER_BLOCK_X 32
#define THREAD_PER_BLOCK_Y 1
#define THREAD_PER_BLOCK_Z 1

#define BLOCK_PER_GRID_X 16
#define BLOCK_PER_GRID_Y 1
#define BLOCK_PER_GRID_Z 1

#define NB_BLOCK  (BLOCK_PER_GRID_X*BLOCK_PER_GRID_Y*BLOCK_PER_GRID_Z)
#define NB_THREAD_PER_BLOCK (THREAD_PER_BLOCK_X*THREAD_PER_BLOCK_Y*THREAD_PER_BLOCK_Z)
#define NB_THREAD (NB_BLOCK*NB_THREAD_PER_BLOCK)

/**
 * http://developer.download.nvidia.com/compute/cuda/4_2/rel/toolkit/docs/online/index.html
 * resumer commande cuda
 */
void pimcCuda(void)
    {
    dim3 blockPerGrid = dim3(BLOCK_PER_GRID_X, BLOCK_PER_GRID_Y, BLOCK_PER_GRID_Z); // ou dim3(1, 1) // mais pas  dim3(1, 1,0)
    dim3 threadPerBlock = dim3(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z); // ou dim3(1,1)

    print(blockPerGrid, threadPerBlock);
    Device::assertDim(blockPerGrid, threadPerBlock);

    int nbDevice = 6;//Device::getDeviceCount();
    //Device::loadCudaDriverDeviceAll();
    for (int i = 0; i < nbDevice; i++)
	Device::loadCudaDriver(i);

#pragma omp parallel for //reduction(+:count) private(x,y,z)
    for (int i = 0; i < nbDevice; i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	int deviceId = 0;
	HANDLE_ERROR(hipGetDevice(&deviceId));
	printf("%d\n",deviceId);

	double nglobal = pow(2, 30);
	int nperThread = nglobal / NB_THREAD;
	//printf("%d\n", nperThread);

	int* ptrHostTab = new int[NB_BLOCK];
	for (int i = 0; i < NB_BLOCK; i++)
	    ptrHostTab[i] = 0;

	int* ptrDevTab = NULL;
	hiprandState* ptrDevtabGeneratorThread = NULL;

	size_t tabsize = NB_BLOCK * sizeof(int);

	HANDLE_ERROR(hipMalloc((void**) &ptrDevtabGeneratorThread, NB_THREAD * sizeof(hiprandState)));

	HANDLE_ERROR(hipMalloc((void**) &ptrDevTab, tabsize));

	kernelGenRandom<<<blockPerGrid,threadPerBlock>>>(ptrDevtabGeneratorThread,deviceId);
	check_CUDA_Error("kernelGenRandom");

	kernelReduction<<<blockPerGrid,threadPerBlock>>>(ptrDevtabGeneratorThread,ptrDevTab,nperThread);
	check_CUDA_Error("kernelReduction");

	HANDLE_ERROR(hipMemcpy(ptrHostTab,ptrDevTab,tabsize,hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipDeviceSynchronize());

	long sum = 0;
	for (int i = 0; i < NB_BLOCK; i++)
	    {
	    sum += ptrHostTab[i];
	    }

	double pi = (double) sum / nglobal * 4;
	cout.precision(15);
	cout << pi << endl;

	HANDLE_ERROR(hipFree(ptrDevtabGeneratorThread));
	HANDLE_ERROR(hipFree(ptrDevTab));
	delete[] ptrHostTab;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__global__ void kernelGenRandom(hiprandState* tabGeneratorThread, int deviceId)
    {
    int tid = Indice1D::tid();

    int deltaSeed = deviceId * 1000000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;

    int seed = 1234 + deltaSeed;
    int sequenceNumber = tid + deltaSequence;
    int offset = deltaOffset;

    hiprand_init(seed, sequenceNumber, offset, &tabGeneratorThread[tid]);

    }

__global__ void kernelReduction(hiprandState* tabGeneratorThread, int* ptrDevTab, int n)
    {
    int tid = Indice1D::tid();
    int tidLocal = threadIdx.x + blockDim.x * threadIdx.y;

    int count = 0;
    hiprandState localState = tabGeneratorThread[tid];

    __shared__ int tabSM[NB_THREAD_PER_BLOCK];
    tabSM[tidLocal] = 0;

    float x, y;
    for (int i = 0; i < n; i++)
	{
	x = hiprand_uniform(&localState);
	y = hiprand_uniform(&localState);
	if (x * x + y * y <= 1)
	    count++;
	}
    tabSM[tidLocal] = count;

    __syncthreads();

    int half = NB_THREAD_PER_BLOCK / 2;
    while (half > 0)
	{
	step(tabSM, half);
	half /= 2;
	}
    copySMtoGM(ptrDevTab, tabSM, n);

    }

__device__ void step(int* tab, int half)
    {
    int tidLocal = threadIdx.x + blockDim.x * threadIdx.y;
    int tabindex = tidLocal;
    if (tidLocal < half)
	{
	tab[tabindex] += tab[tabindex + half];
	}
    __syncthreads();
    }

__device__ void copyGMtoSM(int* tabSM, int* tabGM, int n)
    {
    int tidLocal = threadIdx.x + blockDim.x * threadIdx.y;
    if (tidLocal < n)
	{
	tabSM[tidLocal] = tabGM[tidLocal];
	}
    __syncthreads();
    }

__device__ void copySMtoGM(int* tabGM, int* tabSM, int n)
    {
    int tidLocal = threadIdx.x + blockDim.x * threadIdx.y;
    int bid = (blockIdx.y * gridDim.x) + blockIdx.x;
    if (tidLocal == 0)
	{
	tabGM[bid] = tabSM[0];
	}
    __syncthreads();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

