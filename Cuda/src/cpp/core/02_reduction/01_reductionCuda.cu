#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include "cudaTools.h"
#include "Device.h"
#include "Indice2D.h"
#include <stdio.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

void reductionCuda(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void kernelReduction(float* ptrDevTab, int n);
__device__ static void step(float* tab, int half);
__device__ void copyGMtoSM(float* tabSM, float* tabGM, int n);
__device__ void copySMtoGM(float* tabGM, float* tabSM, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * http://developer.download.nvidia.com/compute/cuda/4_2/rel/toolkit/docs/online/index.html
 * resumer commande cuda
 */
void reductionCuda(void)
    {
    dim3 blockPerGrid = dim3(16, 1, 1); // ou dim3(1, 1) // mais pas  dim3(1, 1,0)
    dim3 threadPerBlock = dim3(32, 1, 1); // ou dim3(1,1)

    print(blockPerGrid, threadPerBlock);
    Device::assertDim(blockPerGrid, threadPerBlock);

    int n = 8;

    float* ptrHostTab = new float[n];
    float* ptrDevTab = NULL;
    for (int i = 0; i < n; i++)
	ptrHostTab[i] = 2;

    printf("input data host side: ");
    for (int i = 0; i < n; i++)
	printf("%f,", ptrHostTab[i]);
    printf("\n");

    size_t tabsize = n * sizeof(float);

    HANDLE_ERROR(hipMalloc((void**) &ptrDevTab, tabsize));
    HANDLE_ERROR(hipMemcpy(ptrDevTab,ptrHostTab,tabsize,hipMemcpyHostToDevice));

    kernelReduction<<<blockPerGrid,threadPerBlock>>>(ptrDevTab,n);

    check_CUDA_Error("kernelReduction");

    HANDLE_ERROR(hipMemcpy(ptrHostTab,ptrDevTab,tabsize,hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipDeviceSynchronize());

    printf("output data host side: ");
    for (int i = 0; i < n; i++)
	printf("%f,", ptrHostTab[i]);
    printf("\n");

    HANDLE_ERROR(hipFree(ptrDevTab));
    delete[] ptrHostTab;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void kernelReduction(float* ptrDevTab, int n)
    {
    int tid = Indice2D::tid();
    int tidLocal = Indice2D::tidLocalBlock();

    __shared__ float tabSM[10];
    tabSM[tidLocal] = 0;

    copyGMtoSM(tabSM, ptrDevTab, n);
    int half = n / 2;
    while (half > 0)
	{
	step(tabSM, half);
	half /= 2;
	}
    copySMtoGM(ptrDevTab, tabSM, n);
    }

__device__ void step(float* tab, int half)
    {
    int tidLocal = Indice2D::tidLocalBlock();
    int tabindex = tidLocal;
    if (tidLocal < half)
	{
	tab[tabindex] += tab[tabindex + half];
	}
    __syncthreads();
    }

__device__ void copyGMtoSM(float* tabSM, float* tabGM, int n)
    {
    int tidLocal = Indice2D::tidLocalBlock();
    if (tidLocal < n)
	{
	tabSM[tidLocal] = tabGM[tidLocal];
	}
    __syncthreads();
    }

__device__ void copySMtoGM(float* tabGM, float* tabSM, int n)
    {
    int tidLocal = Indice2D::tidLocalBlock();
    int bid = (blockIdx.y * gridDim.x) + blockIdx.x;
    if (tidLocal == 0)
	{
	tabGM[bid] = tabSM[0];
	}
    __syncthreads();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

