#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include <iostream>

// TODO: Check DMA memory from capture
/*--------------------------------------*\
 |*   UTILS                             *|
 \*-------------------------------------*/
typedef bool (*compare) ( const uint8_t val1, const uint8_t val2 );

__device__ bool min ( const uint8_t val1, const uint8_t val2 ) {
  return val1 > val2;
}

__device__ bool max ( const uint8_t val1, const uint8_t val2 ) {
  return val1 < val2;
}

/**
 * Computes min-or-max using comparaison function.
 *
 * @param val1 the first value to compare
 * @param val2 the second value to compare
 * @param comp the function used to compare
 * @param result the result from min-or-max computation
 */
__device__ void minOrMax ( const uint8_t val1, const uint8_t val2, const compare comp, uint8_t* result ) {
  *result = val1;
  if ( comp ( val1, val2 ) ) {
    *result = val2;
  }
}

/**
 * Computes Min-or-Max from RGB channels.
 *
 * @param r red channel
 * @param g green channel
 * @param b blue channel
 * @param comp compare function
 * @param result resulting value from comparaison
 */
__device__ void minOrMax3 ( const uint8_t r, const uint8_t g, const uint8_t b, const compare comp, uint8_t* result ) {
  uint8_t resultrg, resultgb;
  minOrMax ( r, g, comp, &resultrg );
  minOrMax ( g, b, comp, &resultgb );
  minOrMax ( resultrg, resultgb, comp, result );
}

/*--------------------------------------*\
 |*   GPU Globals                       *|
 \*-------------------------------------*/
texture<uint8_t, 2, hipReadModeElementType> texBWImage;

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses lightness algorithm:
 * <pre>
 * bw = (max(R, G, B) + min(R, G, B)) / 2
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Lightness ( const uchar4* ptrDevRGBImage, const uint32_t w, const uint32_t h, uint8_t* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    uint8_t minresult, maxresult;
    minOrMax3 ( ptrDevRGBImage[s].x, ptrDevRGBImage[s].y, ptrDevRGBImage[s].z, max, &maxresult );
    minOrMax3 ( ptrDevRGBImage[s].x, ptrDevRGBImage[s].y, ptrDevRGBImage[s].z, min, &minresult );
    uint8_t gray = ( maxresult + minresult ) / 2;
    ptrDevBWImage[s] = gray;
    s += nbThreads;
  }
}

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses average algorithm:
 * <pre>
 * bw = (R + G + B) / 3
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Average ( const uchar4* ptrDevRGBImage, const uint32_t w, const uint32_t h, uint8_t* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    uint8_t gray = ( ptrDevRGBImage[s].x + ptrDevRGBImage[s].y + ptrDevRGBImage[s].z ) / 3;
    ptrDevBWImage[s] = gray;
    s += nbThreads;
  }
}

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses lightness algorithm:
 * <pre>
 * bw = (0.21 * R) + (0.71 * G) + (0.07 * B)
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Luminance ( const uchar4* ptrDevRGBImage, const uint32_t w, const uint32_t h, uint8_t* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  const float R_FACTOR = 0.21f;
  const float G_FACTOR = 0.71f;
  const float B_FACTOR = 0.07f;
  while ( s < size ) {
    uint8_t gray = ( ptrDevRGBImage[s].x * R_FACTOR ) + ( ptrDevRGBImage[s].y * G_FACTOR ) + ( ptrDevRGBImage[s].z * B_FACTOR );
    ptrDevBWImage[s] = gray;
    s += nbThreads;
  }
}

/**
 * Displays given black&white image to the OpenGL context.
 *
 * @param ptrDevBWImage black&white image resulting from convolution
 * @param w width of the image
 * @param h heigth of the image
 * @param ptrDevImageGL OpenGL context image
 */
__global__ void kernelDisplayBWImage ( const uint8_t* ptrDevBWImage, const uint32_t w, const uint32_t h, uchar4* ptrDevImageGL ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    ptrDevImageGL[s].w = 255;
    ptrDevImageGL[s].x = ptrDevImageGL[s].y = ptrDevImageGL[s].z = ptrDevBWImage[s];
    s += nbThreads;
  }
}

/**
 * Convolution function using texture.
 *
 * @param kernel the kernel used for convolution
 * @param k number of column from the kernel
 * @param center center point of the kernel
 * @param kHalf half of the kernel column size
 * @param i i-th position of the image
 * @param j j-th position of the image
 */
__device__ float convolutionKernelTexture ( const float* ptrDevKernel, const uint32_t k, const uint32_t center, const uint32_t kHalf,
    const uint32_t i, const uint32_t j ) {
  float SE, SO, NE, NO;
  float CH, CV;
  for ( uint32_t u = 1; u <= kHalf; u++ ) {
    for ( uint32_t v = 1; v <= kHalf; v++ ) {
      SE += ptrDevKernel[( center + k * v ) + u] * tex2D ( texBWImage, i + v, j + u );
      SO += ptrDevKernel[( center + k * v ) - u] * tex2D ( texBWImage, i + v, j - u );
      NE += ptrDevKernel[( center - k * v ) + u] * tex2D ( texBWImage, i - v, j + u );
      NO += ptrDevKernel[( center - k * v ) - u] * tex2D ( texBWImage, i - v, j - u );
    }
  }
  for ( int u = -( kHalf ); u < ( kHalf ); u++ ) {
    CH += ptrDevKernel[center + u] * tex2D ( texBWImage, i, j + u );
    CV += ptrDevKernel[center + k * u] * tex2D ( texBWImage, i + u, j );
  }
  return SE + SO + NE + NO + CH + CV - ( ptrDevKernel[center] * tex2D ( texBWImage, i, j ) );
}

/**
 * Kernel which compute convolution using texture.
 *
 * @param w width of the image
 * @param h heigth of the image
 * @param kernel kernel used for the convolution
 * @param k number of column from the kernel
 * @param
 */
__global__ void kernelConvolutionTexture ( const uint32_t w, const uint32_t h, const float* ptrDevKernel, const uint32_t k,
    uchar4* ptrDevImageGL ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  int kHalf = ( k / 2 );
  int center = ( k * kHalf ) + kHalf;
  float convolution;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    convolution = convolutionKernelTexture ( ptrDevKernel, k, center, kHalf, i, j );
    ptrDevImageGL[s].w = 255;
    ptrDevImageGL[s].x = ptrDevImageGL[s].y = ptrDevImageGL[s].z = (unsigned char) ( convolution + 64.0f );
    s += nbThreads;
  }
}

/*--------------------------------------*\
 |*   CPU Globals                       *|
 \*-------------------------------------*/
struct CudaImagesSizes {
  uint32_t w;
  uint32_t h;
  size_t rgb_pitch;
  size_t rgb_size;
  size_t bw_pitch;
  size_t bw_size;
  size_t kernel_size;
};
static uchar4* ptrDevCudaRGBImage = NULL;
static uint8_t* ptrDevCudaBWImage = NULL;
static float* ptrDevKernel = NULL;
static CudaImagesSizes sizes;

/**
 * Initializes memory and everythings
 *
 * @param w width of the image
 * @param h heigth of the image
 */
void initKernelFillImage ( const uint32_t w, const uint32_t h, const float* kernel, const size_t kernelSize ) {
  size_t rgb_size = sizeof(uchar4) * h * w;
  size_t bw_size = sizeof(uint8_t) * h * w;
  sizes.w = w;
  sizes.h = h;
  sizes.rgb_pitch = sizeof(uchar4) * w;
  sizes.rgb_size = rgb_size;
  sizes.bw_pitch = sizeof(uint8_t) * w;
  sizes.bw_size = bw_size;
  sizes.kernel_size = kernelSize;
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaRGBImage, rgb_size) );
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaBWImage, bw_size ) );
  HANDLE_ERROR( hipMalloc((void**) &ptrDevKernel, kernelSize) );

  // Copy kernel to global memory
  HANDLE_ERROR( hipMemcpy( ptrDevKernel, kernel, sizes.kernel_size, hipMemcpyHostToDevice ) );

  // Create tex, bind tex to ptrDevCudaBWImage
  texBWImage.addressMode[0] = hipAddressModeClamp;
  texBWImage.addressMode[1] = hipAddressModeClamp;
  texBWImage.filterMode = hipFilterModePoint;
  texBWImage.normalized = false; // coordinate not in [0, 1]
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint8_t> ();
  HANDLE_ERROR( hipBindTexture2D(NULL, texBWImage, ptrDevCudaBWImage, channelDesc, w, h, sizes.bw_pitch ) );
}

/**
 *
 */
void freeKernelFillImageKernel () {

}

/**
 * Launches kernel which fill image.
 * Kernel called are:
 * <ul>
 * <ol>RGB to BW kernel (3 algorithms)</ol>
 * <ol>Convolution kernel (different memory usage)</ol>
 * </ul>
 *
 * TODO:
 * - Histogram from image
 * - Frangi Filter impl. in CUDA
 *
 * @param ptrDevImageGL OpenGL context image (DEVICE)
 * @param ptrCudaImage image read from video (HOST)
 * @param w width of the image
 * @param h heigth of the image
 */
void launchKernelFillImageKernel ( uchar4* ptrDevImageGL, const uchar4* ptrCudaImage, const uint32_t w, const uint32_t h ) {
  HANDLE_ERROR( hipMemcpy( ptrDevCudaRGBImage, ptrCudaImage, sizes.rgb_size, hipMemcpyHostToDevice ) );
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
  kernelRGBImageToBW_Lightness<<< dg, db >>> ( ptrDevCudaRGBImage, w, h, ptrDevCudaBWImage );
  //HANDLE_ERROR( hipDeviceSynchronize() );
  kernelConvolutionTexture<<< dg, db >>> ( w, h, ptrDevKernel, 9, ptrDevImageGL );
  //kernelDisplayBWImage<<< dg, db >>> ( ptrDevCudaBWImage, w, h, ptrDevImageGL );
  // TODO: Use switch to launch memory cases
  ;//
}
