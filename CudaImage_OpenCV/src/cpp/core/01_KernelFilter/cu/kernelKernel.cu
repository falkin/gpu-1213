#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "KernelFilterImpl.h"
#include "Chronos.h"
#include <iostream>

/*--------------------------------------*\
 |*   UTILS                             *|
 \*-------------------------------------*/
typedef bool (*compare) ( const uint8_t val1, const uint8_t val2 );

__device__ bool min ( const uint8_t val1, const uint8_t val2 ) {
  return val1 > val2;
}

__device__ bool max ( const uint8_t val1, const uint8_t val2 ) {
  return val1 < val2;
}

/**
 * Computes min-or-max using comparaison function.
 *
 * @param val1 the first value to compare
 * @param val2 the second value to compare
 * @param comp the function used to compare
 * @param result the result from min-or-max computation
 */
__device__ void minOrMax ( const uint8_t val1, const uint8_t val2, const compare comp, uint8_t* result ) {
  *result = val1;
  if ( comp ( val1, val2 ) ) {
    *result = val2;
  }
}

/**
 * Computes Min-or-Max from RGB channels.
 *
 * @param r red channel
 * @param g green channel
 * @param b blue channel
 * @param comp compare function
 * @param result resulting value from comparaison
 */
__device__ void minOrMax3 ( const uint8_t r, const uint8_t g, const uint8_t b, const compare comp, uint8_t* result ) {
  uint8_t resultrg, resultgb;
  minOrMax ( r, g, comp, &resultrg );
  minOrMax ( g, b, comp, &resultgb );
  minOrMax ( resultrg, resultgb, comp, result );
}

/*--------------------------------------*\
 |*   GPU Globals                       *|
 \*-------------------------------------*/
texture<uchar4, 2, hipReadModeElementType> texBWImage;
__constant__ float k_KERNEL[81];

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses lightness algorithm:
 * <pre>
 * bw = (max(R, G, B) + min(R, G, B)) / 2
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Lightness ( const uchar4* ptrDevRGBImage, const uint32_t w, const uint32_t h, uchar4* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    uint8_t minresult, maxresult;
    minOrMax3 ( ptrDevRGBImage[s].x, ptrDevRGBImage[s].y, ptrDevRGBImage[s].z, max, &maxresult );
    minOrMax3 ( ptrDevRGBImage[s].x, ptrDevRGBImage[s].y, ptrDevRGBImage[s].z, min, &minresult );
    uint8_t gray = ( maxresult + minresult ) / 2;
    ptrDevBWImage[s].x = ptrDevBWImage[s].y = ptrDevBWImage[s].z = ptrDevBWImage[s].w = gray;
    s += nbThreads;
  }
}

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses average algorithm:
 * <pre>
 * bw = (R + G + B) / 3
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Average ( const uchar4* ptrDevRGBImage, const uint32_t w, const uint32_t h, uchar4* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    uint8_t gray = ( ptrDevRGBImage[s].x + ptrDevRGBImage[s].y + ptrDevRGBImage[s].z ) / 3;
    ptrDevBWImage[s].x = ptrDevBWImage[s].y = ptrDevBWImage[s].z = ptrDevBWImage[s].w = gray;
    s += nbThreads;
  }
}

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses lightness algorithm:
 * <pre>
 * bw = (0.21 * R) + (0.71 * G) + (0.07 * B)
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Luminance ( const uchar4* ptrDevRGBImage, const uint32_t w, const uint32_t h, uchar4* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  const float R_FACTOR = 0.21f;
  const float G_FACTOR = 0.71f;
  const float B_FACTOR = 0.07f;
  while ( s < size ) {
    uint8_t gray = ( ptrDevRGBImage[s].x * R_FACTOR ) + ( ptrDevRGBImage[s].y * G_FACTOR ) + ( ptrDevRGBImage[s].z * B_FACTOR );
    ptrDevBWImage[s].x = ptrDevBWImage[s].y = ptrDevBWImage[s].z = ptrDevBWImage[s].w = gray;
    s += nbThreads;
  }
}

/**
 * Convolution function using texture for the image and global memory for the kernel.
 *
 * @param kernel the kernel used for convolution
 * @param k number of column from the kernel
 * @param center center point of the kernel
 * @param kHalf half of the kernel column size
 * @param i i-th position of the image
 * @param j j-th position of the image
 */
__device__ float convolutionKernelGlobalImageTexture ( const float* ptrDevKernel, const uint32_t k, const uint32_t center,
    const uint32_t kHalf, const uint32_t i, const uint32_t j ) {
  float sum = 0.0f;
  for ( int v = 1; v <= kHalf; v++ ) {
    for ( int u = 1; u <= kHalf; u++ ) {
      sum += ptrDevKernel[center + ( v * k ) + u] * tex2D ( texBWImage, j + v, i + u ).x;
      sum += ptrDevKernel[center + ( v * k ) - u] * tex2D ( texBWImage, j + v, i - u ).x;
      sum += ptrDevKernel[center - ( v * k ) + u] * tex2D ( texBWImage, j - v, i + u ).x;
      sum += ptrDevKernel[center - ( v * k ) - u] * tex2D ( texBWImage, j - v, i - u ).x;
    }
  }
  for ( int u = -k / 2; u < k / 2; u++ ) {
    sum += ptrDevKernel[center + u] * tex2D ( texBWImage, j, i + u ).x;
    sum += ptrDevKernel[center + k * u] * tex2D ( texBWImage, j + u, i ).x;
  }
  sum += ( ptrDevKernel[center] * tex2D ( texBWImage, j, i ).x ); // Center computed twice
  return sum;
}

/**
 * Kernel stored in global memory which compute convolution using texture mapping to the image.
 *
 * @param w width of the image
 * @param h heigth of the image
 * @param kernel kernel used for the convolution
 * @param k number of column from the kernel
 * @param ptrDevCudaImageConvolution the convolution result
 */
__global__ void kernelConvolutionGlobalImageTexture ( const uint32_t w, const uint32_t h, const float* ptrDevKernel, const uint32_t k,
    float* ptrDevCudaImageConvolution ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  int kHalf = ( k / 2 );
  int center = k * ( k / 2 ) + kHalf;
  float convolution;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    convolution = convolutionKernelGlobalImageTexture ( ptrDevKernel, k, center, kHalf, i, j );
    ptrDevCudaImageConvolution[s] = convolution;
    s += nbThreads;
  }
}

/**
 * Convolution function using texture for the image and constant memory for the kernel.
 *
 * @param kernel the kernel used for convolution
 * @param k number of column from the kernel
 * @param center center point of the kernel
 * @param kHalf half of the kernel column size
 * @param i i-th position of the image
 * @param j j-th position of the image
 */
__device__ float convolutionKernelConstantImageTexture ( const uint32_t k, const uint32_t center, const uint32_t kHalf, const uint32_t i,
    const uint32_t j ) {
  float sum = 0.0f;
  for ( int v = 1; v <= kHalf; v++ ) {
    for ( int u = 1; u <= kHalf; u++ ) {
      sum += k_KERNEL[center + ( v * k ) + u] * tex2D ( texBWImage, j + v, i + u ).x;
      sum += k_KERNEL[center + ( v * k ) - u] * tex2D ( texBWImage, j + v, i - u ).x;
      sum += k_KERNEL[center - ( v * k ) + u] * tex2D ( texBWImage, j - v, i + u ).x;
      sum += k_KERNEL[center - ( v * k ) - u] * tex2D ( texBWImage, j - v, i - u ).x;
    }
  }
  for ( int u = -k / 2; u < k / 2; u++ ) {
    sum += k_KERNEL[center + u] * tex2D ( texBWImage, j, i + u ).x;
    sum += k_KERNEL[center + k * u] * tex2D ( texBWImage, j + u, i ).x;
  }
  sum += ( k_KERNEL[center] * tex2D ( texBWImage, j, i ).x ); // Center computed twice.
  return sum;
}

/**
 * Kernel stored in global memory which compute convolution using texture mapping to the image.
 *
 * @param w width of the image
 * @param h heigth of the image
 * @param kernel kernel used for the convolution
 * @param k number of column from the kernel
 * @param ptrDevCudaImageConvolution the convolution result
 */
__global__ void kernelConvolutionConstantImageTexture ( const uint32_t w, const uint32_t h, const uint32_t k,
    float* ptrDevCudaImageConvolution ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  int kHalf = ( k / 2 );
  int center = k * ( k / 2 ) + kHalf;
  float convolution;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    convolution = convolutionKernelConstantImageTexture ( k, center, kHalf, i, j );
    ptrDevCudaImageConvolution[s] = convolution;
    s += nbThreads;
  }
}

/**
 * Convolution function using texture for the image and constant memory for the kernel.
 *
 * @param kernel the kernel used for convolution
 * @param k number of column from the kernel
 * @param center center point of the kernel
 * @param kHalf half of the kernel column size
 * @param i i-th position of the image
 * @param j j-th position of the image
 */
__device__ float convolutionKernelConstantImageTexture_mul24 ( const uint32_t k, const uint32_t center, const uint32_t kHalf,
    const uint32_t i, const uint32_t j ) {
  float sum = 0.0f;
  for ( int v = 1; v <= kHalf; v++ ) {
    for ( int u = 1; u <= kHalf; u++ ) {
      sum += __mul24 ( k_KERNEL[center + ( v * k ) + u], tex2D ( texBWImage, j + v, i + u ).x );
      sum += __mul24 ( k_KERNEL[center + ( v * k ) - u], tex2D ( texBWImage, j + v, i - u ).x );
      sum += __mul24 ( k_KERNEL[center - ( v * k ) + u], tex2D ( texBWImage, j - v, i + u ).x );
      sum += __mul24 ( k_KERNEL[center - ( v * k ) - u], tex2D ( texBWImage, j - v, i - u ).x );
    }
  }
  for ( int u = -k / 2; u < k / 2; u++ ) {
    sum += __mul24 ( k_KERNEL[center + u], tex2D ( texBWImage, j, i + u ).x );
    sum += __mul24 ( k_KERNEL[center + k * u], tex2D ( texBWImage, j + u, i ).x );
  }
  sum += ( __mul24 ( k_KERNEL[center], tex2D ( texBWImage, j, i ).x ) ); // Center computed twice.
  return sum;
}

/**
 * Kernel stored in global memory which compute convolution using texture mapping to the image.
 *
 * @param w width of the image
 * @param h heigth of the image
 * @param kernel kernel used for the convolution
 * @param k number of column from the kernel
 * @param ptrDevCudaImageConvolution the convolution result
 */
__global__ void kernelConvolutionConstantImageTexture_mul24 ( const uint32_t w, const uint32_t h, const uint32_t k,
    float* ptrDevCudaImageConvolution ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  int kHalf = ( k / 2 );
  int center = k * ( k / 2 ) + kHalf;
  float convolution;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    convolution = convolutionKernelConstantImageTexture ( k, center, kHalf, i, j );
    ptrDevCudaImageConvolution[s] = convolution;
    s += nbThreads;
  }
}

/**
 * Displays given black&white image to the OpenGL context.
 *
 * @param ptrDevConvolution image resulting from convolution
 * @param w width of the image
 * @param h heigth of the image
 * @param ptrDevImageGL OpenGL context image
 */
__global__ void kernelDisplayConvolution ( const float* ptrDevConvolution, const uint32_t w, const uint32_t h, uchar4* ptrDevImageGL ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    ptrDevImageGL[s].w = 255;
    ptrDevImageGL[s].x = ptrDevImageGL[s].y = ptrDevImageGL[s].z = (uint8_t) ( ptrDevConvolution[s] );
    s += nbThreads;
  }
}

/*--------------------------------------*\
 |*   CPU Globals                       *|
 \*-------------------------------------*/
struct CudaImagesSizes {
  uint32_t w;
  uint32_t h;
  size_t rgb_pitch;
  size_t rgb_size;
  size_t bw_pitch;
  size_t bw_size;
  size_t conv_size;
  size_t kernel_size;
};
static uchar4* ptrDevCudaRGBImage = NULL;
static uchar4* ptrDevCudaBWImage = NULL;
static float* ptrDevKernel = NULL;
static float* ptrDevCudaImageConvolution = NULL;
static CudaImagesSizes sizes;

/**
 * Initializes memory and everythings
 *
 * @param w width of the image
 * @param h heigth of the image
 */
void initKernelFillImage ( const uint32_t w, const uint32_t h, const float* kernel, const size_t kernelSize ) {
  size_t rgb_size = sizeof(uchar4) * h * w;
  size_t bw_size = sizeof(uchar4) * h * w;
  size_t conv_size = sizeof(float) * h * w;
  sizes.w = w;
  sizes.h = h;
  sizes.rgb_pitch = sizeof(uchar4) * w;
  sizes.rgb_size = rgb_size;
  sizes.bw_pitch = sizeof(uchar4) * w;
  sizes.bw_size = bw_size;
  sizes.conv_size = conv_size;
  sizes.kernel_size = kernelSize * sizeof(float);
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaRGBImage, rgb_size) );
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaBWImage, bw_size ) );
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaImageConvolution, conv_size ) );
  HANDLE_ERROR( hipMalloc((void**) &ptrDevKernel, sizes.kernel_size) );

// Copy kernel to global memory
  HANDLE_ERROR( hipMemcpy( ptrDevKernel, kernel, sizes.kernel_size, hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("k_KERNEL"), kernel, sizes.kernel_size, 0, hipMemcpyHostToDevice ) );

// Create tex, bind tex to ptrDevCudaBWImage
  texBWImage.addressMode[0] = hipAddressModeWrap;
  texBWImage.addressMode[1] = hipAddressModeWrap;
  //texBWImage.filterMode = hipFilterModePoint;
  texBWImage.normalized = false; // coordinate not in [0, 1]
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4> ();
  HANDLE_ERROR( hipBindTexture2D(NULL, &texBWImage, ptrDevCudaBWImage, &channelDesc, w, h, sizes.bw_pitch ) );
}

/**
 *
 */
void freeKernelFillImageKernel () {
  //HANDLE_ERROR (  );
}

/**
 * Launches kernel which fill image.
 * Kernel called are:
 * <ul>
 * <ol>RGB to BW kernel (3 algorithms)</ol>
 * <ol>Convolution kernel (different memory usage)</ol>
 * </ul>
 *
 * @param ptrDevImageGL OpenGL context image (DEVICE)
 * @param ptrCudaImage image read from video (HOST)
 * @param w width of the image
 * @param h heigth of the image
 */
double launchKernelFillImageKernel ( uchar4* ptrDevImageGL, const uchar4* ptrCudaImage, const uint32_t w, const uint32_t h,
    const KernelKind kind, const GrayscaleMethod grayscale, const dim3 dg, const dim3 db ) {
  Chronos chrono;
  HANDLE_ERROR( hipMemcpy( ptrDevCudaRGBImage, ptrCudaImage, sizes.rgb_size, hipMemcpyHostToDevice ) );
  chrono.start ();
  switch ( grayscale ) {
  default:
  case AVERAGE:
    kernelRGBImageToBW_Average<<< dg, db >>> ( ptrDevCudaRGBImage, w, h, ptrDevCudaBWImage );
    break;
  case LIGHTNESS:
    kernelRGBImageToBW_Lightness<<< dg, db >>> ( ptrDevCudaRGBImage, w, h, ptrDevCudaBWImage );
    break;
  case LUMINANCE:
    kernelRGBImageToBW_Luminance<<< dg, db >>> ( ptrDevCudaRGBImage, w, h, ptrDevCudaBWImage );
    break;
  }
  switch ( kind ) {
  default:
  case TEXTURE_GLOBAL:
    kernelConvolutionGlobalImageTexture<<< dg, db >>> ( w, h, ptrDevKernel, 9, ptrDevCudaImageConvolution );
    break;
  case TEXTURE_CONSTANT:
    kernelConvolutionConstantImageTexture<<< dg, db >>> ( w, h, 9, ptrDevCudaImageConvolution );
    break;
  case TEXTURE_CONSTANT_MUL24:
    kernelConvolutionConstantImageTexture_mul24<<< dg, db >>> ( w, h, 9, ptrDevCudaImageConvolution );
    break;
  }
  HANDLE_ERROR( hipDeviceSynchronize() );
  double time = chrono.stop ();
  kernelDisplayConvolution<<< dg, db >>> ( ptrDevCudaImageConvolution, w, h, ptrDevImageGL );
  ; //
  return time;
}
