#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include <iostream>

// TODO: Check DMA memory from capture
/*--------------------------------------*\
 |*   UTILS                             *|
 \*-------------------------------------*/
typedef bool (*compare) ( const unsigned char val1, const unsigned char val2 );

__device__ bool min ( const unsigned char val1, const unsigned char val2 ) {
  return val1 > val2;
}

__device__ bool max ( const unsigned char val1, const unsigned char val2 ) {
  return val1 < val2;
}

/**
 * Computes min-or-max using comparaison function.
 *
 * @param val1 the first value to compare
 * @param val2 the second value to compare
 * @param comp the function used to compare
 * @param result the result from min-or-max computation
 */
__device__ void minOrMax ( const unsigned char val1, const unsigned char val2, const compare comp, unsigned char* result ) {
  *result = val1;
  if ( comp ( val1, val2 ) ) {
    *result = val2;
  }
}

/**
 * Computes Min-or-Max from RGB channels.
 *
 * @param r red channel
 * @param g green channel
 * @param b blue channel
 * @param comp compare function
 * @param result resulting value from comparaison
 */
__device__ void minOrMax3 ( const unsigned char r, const unsigned char g, const unsigned char b, const compare comp,
    unsigned char* result ) {
  unsigned char resultrg, resultgb;
  minOrMax ( r, g, comp, &resultrg );
  minOrMax ( g, b, comp, &resultgb );
  minOrMax ( resultrg, resultgb, comp, result );
}

/*--------------------------------------*\
 |*   GPU Globals                       *|
 \*-------------------------------------*/
texture<unsigned char, 2, hipReadModeElementType> texBWImage;

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses lightness algorithm:
 * <pre>
 * bw = (max(R, G, B) + min(R, G, B)) / 2
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Lightness ( const uchar4* ptrDevRGBImage, const unsigned int w, const unsigned int h,
    unsigned char* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    unsigned char minresult, maxresult;
    minOrMax3 ( ptrDevRGBImage[s].x, ptrDevRGBImage[s].y, ptrDevRGBImage[s].z, max, &maxresult );
    minOrMax3 ( ptrDevRGBImage[s].x, ptrDevRGBImage[s].y, ptrDevRGBImage[s].z, min, &minresult );
    unsigned char gray = ( maxresult + minresult ) / 2;
    ptrDevBWImage[s] = gray;
    s += nbThreads;
  }
}

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses average algorithm:
 * <pre>
 * bw = (R + G + B) / 3
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Average ( const uchar4* ptrDevRGBImage, const unsigned int w, const unsigned int h,
    unsigned char* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    unsigned char gray = ( ptrDevRGBImage[s].x + ptrDevRGBImage[s].y + ptrDevRGBImage[s].z ) / 3;
    ptrDevBWImage[s] = gray;
    s += nbThreads;
  }
}

/**
 * Computes Grayscale image from RGB image.
 * <p> Uses lightness algorithm:
 * <pre>
 * bw = (0.21 * R) + (0.71 * G) + (0.07 * B)
 * </pre>
 *
 * @param ptrDevRGBImage RGB image read from video
 * @param w width of the image
 * @param h height of the image
 * @param ptrDevBWImage black&white image computed
 */
__global__ void kernelRGBImageToBW_Luminance ( const uchar4* ptrDevRGBImage, const unsigned int w, const unsigned int h,
    unsigned char* ptrDevBWImage ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  const float R_FACTOR = 0.21f;
  const float G_FACTOR = 0.71f;
  const float B_FACTOR = 0.07f;
  while ( s < size ) {
    unsigned char gray = ( ptrDevRGBImage[s].x * R_FACTOR ) + ( ptrDevRGBImage[s].y * G_FACTOR ) + ( ptrDevRGBImage[s].z * B_FACTOR );
    ptrDevBWImage[s] = gray;
    s += nbThreads;
  }
}

/**
 * Displays given black&white image to the OpenGL context.
 *
 * @param ptrDevBWImage black&white image resulting from convolution
 * @param w width of the image
 * @param h heigth of the image
 * @param ptrDevImageGL OpenGL context image
 */
__global__ void kernelDisplayBWImage ( const unsigned char* ptrDevBWImage, const unsigned int w, const unsigned int h,
    uchar4* ptrDevImageGL ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  while ( s < size ) {
    ptrDevImageGL[s].w = 255;
    ptrDevImageGL[s].x = ptrDevImageGL[s].y = ptrDevImageGL[s].z = ptrDevBWImage[s];
    s += nbThreads;
  }
}

/**
 * Convolution function using texture.
 *
 * @param kernel the kernel used for convolution
 * @param k number of column from the kernel
 * @param center center point of the kernel
 * @param kHalf half of the kernel column size
 * @param i i-th position of the image
 * @param j j-th position of the image
 */
__device__ float convolutionKernelTexture ( const float* kernel, const unsigned int k, const unsigned int center, const unsigned int kHalf,
    const unsigned int i, const unsigned int j ) {
  float SE, SO, NE, NO;
  float CH, CV;
  for ( unsigned int u = 0; u < kHalf; u++ ) {
    for ( unsigned int v = 0; v < kHalf; v++ ) {
      SE += kernel[( center + k * v ) + u] * tex2D ( texBWImage, i + v, j + u );
      SO += kernel[( center + k * v ) - u] * tex2D ( texBWImage, i + v, j - u );
      NE += kernel[( center - k * v ) + u] * tex2D ( texBWImage, i - v, j + u );
      NO += kernel[( center - k * v ) - u] * tex2D ( texBWImage, i - v, j - u );
    }
  }
  for ( int u = -( kHalf ); u < ( kHalf ); u++ ) {
    CH += kernel[center + u] * tex2D ( texBWImage, i, j + u );
    CV += kernel[center + k * u] * tex2D ( texBWImage, i + u, j );
  }
  return SE + SO + NE + NO + CH + CV - ( kernel[center] * tex2D ( texBWImage, i, j ) );
}

/**
 * Kernel which compute convolution using texture.
 *
 * @param w width of the image
 * @param h heigth of the image
 * @param kernel kernel used for the convolution
 * @param k number of column from the kernel
 * @param
 */
__global__ void kernelConvolutionTexture ( const unsigned int w, const unsigned int h, const float* kernel, const unsigned int k,
    uchar4* ptrDevImageGL ) {
  int tid = Indice2D::tid ();
  int nbThreads = Indice2D::nbThread ();
  int s = tid;
  size_t size = h * w;
  int i, j;
  int kHalf = ( k / 2 );
  int center = ( k * kHalf );
  float convolution;
  while ( s < size ) {
    Indice2D::pixelIJ ( s, w, i, j );
    convolution = convolutionKernelTexture ( kernel, k, center, kHalf, i, j );
    ptrDevImageGL[s].w = 255;
    ptrDevImageGL[s].x = ptrDevImageGL[s].y = ptrDevImageGL[s].z = convolution;
    s += nbThreads;
  }
}

/*--------------------------------------*\
 |*   CPU Globals                       *|
 \*-------------------------------------*/
struct CudaImagesSizes {
  unsigned int w;
  unsigned int h;
  size_t rgb_pitch;
  size_t rgb_size;
  size_t bw_pitch;
  size_t bw_size;
};
static uchar4* ptrDevCudaRGBImage = NULL;
static unsigned char* ptrDevCudaBWImage = NULL;
static CudaImagesSizes sizes;

/**
 * Initializes memory and everythings
 *
 * @param w width of the image
 * @param h heigth of the image
 */
void initKernelFillImage ( const unsigned int w, const unsigned int h ) {
  size_t rgb_size = sizeof(uchar4) * h * w;
  size_t bw_size = sizeof(unsigned char) * h * w;
  sizes.w = w;
  sizes.h = h;
  sizes.rgb_pitch = sizeof(uchar4) * w;
  sizes.rgb_size = rgb_size;
  sizes.bw_pitch = sizeof(unsigned char) * w;
  sizes.bw_size = bw_size;
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaRGBImage, rgb_size) );
  HANDLE_ERROR( hipMalloc((void**) &ptrDevCudaBWImage, bw_size ) );
  // Create tex, bind tex to ptrDevCudaBWImage
  texBWImage.addressMode[0] = hipAddressModeClamp;
  texBWImage.addressMode[1] = hipAddressModeClamp;
  texBWImage.filterMode = hipFilterModePoint;
  texBWImage.normalized = false; // coordinate not in [0, 1]
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char> ();
  HANDLE_ERROR( hipBindTexture2D(NULL, texBWImage, ptrDevCudaBWImage, channelDesc, w, h, sizes.bw_pitch ) );
}

void freeKernelFillImageKernel () {

}

/**
 * Launches kernel which fill image.
 * Kernel called are:
 * <ul>
 * <ol>RGB to BW kernel (3 algorithms)</ol>
 * <ol>Convolution kernel (different memory usage)</ol>
 * </ul>
 *
 * TODO:
 * - Histogram from image
 * - Frangi Filter impl. in CUDA
 *
 * @param ptrDevImageGL OpenGL context image (DEVICE)
 * @param ptrCudaImage image read from video (HOST)
 * @param w width of the image
 * @param h heigth of the image
 */
void launchKernelFillImageKernel ( uchar4* ptrDevImageGL, const uchar4* ptrCudaImage, const unsigned int w, const unsigned int h ) {
  HANDLE_ERROR( hipMemcpy( ptrDevCudaRGBImage, ptrCudaImage, sizes.rgb_size, hipMemcpyHostToDevice ) );
  dim3 dg = dim3 ( 16, 1, 1 );
  dim3 db = dim3 ( 32, 1, 1 );
kernelRGBImageToBW_Lightness<<< dg, db >>>(ptrDevCudaRGBImage, w, h, ptrDevCudaBWImage );
//HANDLE_ERROR( hipDeviceSynchronize() );
//kernelDisplayBWImage<<< dg, db >>>(ptrDevCudaBWImage, w, h, ptrDevImageGL);
// TODO:
}
