#include <stdint.h>
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Chronos.h"

double launchKernelMemoryTransfert ( const void* memory, const size_t size ) {
  Chronos chrono;
  void* ptrDevMemory;
  chrono.start ();
  HANDLE_ERROR( hipMalloc( &ptrDevMemory, size ) );
  HANDLE_ERROR( hipMemcpy ( ptrDevMemory, memory, size, hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipDeviceSynchronize () );
  HANDLE_ERROR( hipFree ( ptrDevMemory ) );

  return chrono.timeFlight ();
}
