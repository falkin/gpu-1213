#include <stdint.h>
#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Chronos.h"

double launchKernelMemoryTransfert ( const uint32_t w, const uint32_t h, const void* memory, const size_t memory_size ) {
  Chronos chrono;
  size_t size = w * h * memory_size;
  void* ptrDevMemory;
  chrono.start ();
  HANDLE_ERROR( hipMalloc( &ptrDevMemory, size ) );
  HANDLE_ERROR( hipMemcpy ( ptrDevMemory, memory, size, hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipDeviceSynchronize () );
  HANDLE_ERROR( hipFree ( ptrDevMemory ) );

  return chrono.timeFlight ();
}
